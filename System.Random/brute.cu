#include <iostream>
#include <thread>
#include <ctype.h>
#include <time.h>
#include <string>

#include "hip/hip_runtime.h"

#define MIN     1000000
#define MAX     1000000000
#define TARGET  638101650

__global__ void brute_kernel(int *result, int offset) {
    int seed = blockIdx.x * blockDim.x + threadIdx.x + offset;
    if(seed >= (INT_MAX-1)) {
        return;
    }
    int i, j, s, val, randnum;
    int seedarray[56];
    s = 161803398 - seed;
    seedarray[55] = s;
    i = val = 1;
    while(i < 55) {
        j = 21 * i % 55;
        seedarray[j] = val;
        val = s - val;
        if(val < 0) val += INT_MAX;
        s = seedarray[j];
        i++;
    }
    for(i = 1; i < 4; i++) {
        for(j = 1; j < 56; j++) {
            seedarray[j] -= seedarray[1 + (j + 30) % 55];
            if(seedarray[j] < 0) seedarray[j] += INT_MAX;
        }
    }
    for(j = 1; j < 23; j++) {
        seedarray[j] -= seedarray[1 + (j + 30) % 55];
        if(seedarray[j] < 0) seedarray[j] += INT_MAX;
    }
    randnum = seedarray[1] - seedarray[22];
    if (randnum == INT_MAX) randnum--;
    if (randnum < 0) randnum += INT_MAX;
    double rr = randnum*(1.0 / INT_MAX);

    long range = (long)(MAX - MIN);
    if ((int)(rr * range) + MIN == TARGET) {
        *result = seed;
    }
}

int main(int argc, char* argv[]){
    unsigned long long brute_size = INT_MAX;
    unsigned int brute_blocks = 512, brute_threads = 512;

    int *d_result, *result = (int *)malloc(sizeof(int));
    memset(result, 0, sizeof(int));
    clock_t start_t = clock();
    hipMalloc((void**) &d_result, sizeof(int));

    for(int i = 0; i < (brute_size/(brute_blocks*brute_threads))+1; i++) {
        brute_kernel<<<brute_blocks,brute_threads>>>(
            d_result, i *(brute_blocks*brute_threads));
        hipMemcpy(result, d_result, sizeof(int), hipMemcpyDeviceToHost);
        if(*result != 0){
            std::cout << "Seed found:\t" << *result << std::endl;
            hipFree(d_result);
            clock_t total_t = (clock() - start_t);
            std::cout << "Elapsed Time:\t"
                      << (double)total_t/((double)CLOCKS_PER_SEC)
                      << " seconds" << std::endl;
            return 0;
        }
    }
    hipFree(d_result);
    free(result);
}